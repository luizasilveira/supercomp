#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 

// thrust::device_vector<int> v(5, 0); // vetor de 5 ints zerado
// // v = {0, 0, 0, 0, 0}
// thrust::sequence(v.begin(), v.end()); // preenche com 0, 1, 2, ....
// // v = {0, 1, 2, 3, 4}
// thrust::fill(v.begin(), v.begin()+2, 13); // dois primeiros elementos = 13
// // v = {13, 13, 2, 3, 4}

int main() {
    thrust::device_vector<double> AAPL;
    thrust::device_vector<double> MSFT;
    thrust::device_vector<double> MEAN_DIF(2518,0);
    double stocks_AAPL, stocks_MSFT, mean, var;

    for(int i =0; i < 2518; i++){
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }

    int N = AAPL.size();
    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    //double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(),0,thrust::plus<double>());
    mean = val/2517;
    std::cout << "Média: " << fabs(mean) << "\n";

    thrust::device_vector<double> MV_DIF(N); // vetor de 5 ints zerado
    //thrust::fill(MV_DIF.begin(), MV_DIF.end(), mean); // dois primeiros elementos = 13


    // minus a constant (mean)
    thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), thrust::constant_iterator<double>(mean), MV_DIF.begin(), thrust::minus<double>());
    // thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), MV.begin(), MV_DIF.begin(), thrust::minus<double>());

    // multiplica por ele mesmo (**2)
    thrust::transform(MV_DIF.begin(), MV_DIF.end(), MV_DIF.begin(), MV_DIF.begin(), thrust::multiplies<double>());

    // Somatoria
    //thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(),0,thrust::plus<double>());
    double val2 = thrust::reduce(MV_DIF.begin(), MV_DIF.end(), 0,thrust::plus<double>());
    var = val2/N;
    std::cout << "Variância: " << var << "\n";

}
