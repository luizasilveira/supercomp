#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 

// thrust::device_vector<int> v(5, 0); // vetor de 5 ints zerado
// // v = {0, 0, 0, 0, 0}
// thrust::sequence(v.begin(), v.end()); // preenche com 0, 1, 2, ....
// // v = {0, 1, 2, 3, 4}
// thrust::fill(v.begin(), v.begin()+2, 13); // dois primeiros elementos = 13
// // v = {13, 13, 2, 3, 4}

int main() {
    thrust::device_vector<double> stocks;
    // thrust::device_vector<double> s_dif(2518,0);
    double stock;

    for(int i =0; i < 2518; i++){
        std::cin >> stock;
        stocks.push_back(stock);
        
    }

    thrust::device_vector<double> ganho_diario(stocks.size() -1);

    thrust::transform(stocks.begin() + 1, stocks.end(), stocks.begin(), ganho_diario.begin(), thrust::minus<double>());

    printf("Ganho: ");
    for (thrust::device_vector<double>::iterator i = ganho_diario.begin(); i != ganho_diario.end(); i++) {
        std::cout << *i << " ";
        std::cout << "\n";
    }
    // printf("\n");
    //std::cout << "DIferença: " << s_dif << "\n";

    // int N = AAPL.size();
    // thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    // //double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    // double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(),0,thrust::plus<double>());
    // mean = val/2517;
    // std::cout << "Média: " << fabs(mean) << "\n";

    // thrust::device_vector<double> MV_DIF(N); // vetor de 5 ints zerado
    // //thrust::fill(MV_DIF.begin(), MV_DIF.end(), mean); // dois primeiros elementos = 13


    // // minus a constant (mean)
    // thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), thrust::constant_iterator<double>(mean), MV_DIF.begin(), thrust::minus<double>());
    // // thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), MV.begin(), MV_DIF.begin(), thrust::minus<double>());

    // // multiplica por ele mesmo (**2)
    // thrust::transform(MV_DIF.begin(), MV_DIF.end(), MV_DIF.begin(), MV_DIF.begin(), thrust::multiplies<double>());

    // // Somatoria
    // //thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(),0,thrust::plus<double>());
    // double val2 = thrust::reduce(MV_DIF.begin(), MV_DIF.end(), 0,thrust::plus<double>());
    // var = val2/N;
    // std::cout << "Variância: " << var << "\n";

}
