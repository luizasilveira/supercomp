#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 

int main() {
    thrust::device_vector<double> AAPL;
    thrust::device_vector<double> MSFT;
    thrust::device_vector<double> MEAN_DIF(2518,0);
    double stocks_AAPL, stocks_MSFT, mean;

    for(int i =0; i < 2518; i++){
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }


    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    //double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(),0,thrust::plus<double>());
    mean = val/2517;
    std::cout << "Média: " << fabs(mean) << "\n";

    // printf("V1: ");
    // for (thrust::device_vector<double>::iterator i = MEAN_DIF.begin(); i != MEAN_DIF.end(); i++) {
    //     std::cout << *i << " ";
    // }
    // printf("\n");

    // printf("V2: ");
    // for (thrust::device_vector<double>::iterator i = V2.begin(); i != V2.end(); i++) {
    //     std::cout << *i << " ";
    // }
    // printf("\n");

    // printf("V3: ");
    // for (thrust::device_vector<double>::iterator i = V3.begin(); i != V3.end(); i++) {
    //     std::cout << *i << " ";
    // }
    // printf("\n");

    // printf("V4: ");
    // for (thrust::device_vector<double>::iterator i = V4.begin(); i != V4.end(); i++) {
    //     std::cout << *i << " ";
    // }
    // printf("\n");

}
